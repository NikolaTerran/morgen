#include "hip/hip_runtime.h"
#include "engine.h"

//using cuda to initiate canvas array

__global__ void gp_array_list_init(int *list, int val){

	list[blockIdx.x] = val;
	//printf("list[%d]:%d\n",blockIdx.x,list[blockIdx.x]);
}

__global__ void gp_array_set(int *d_r,int *d_g,int *d_b, int *d_x, int *d_y, int r, int g, int b){//int *d_r, int *d_g, int *d_b, int *x, int *y){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int x_lim = X_MAX - X_MIN;

	printf("d_r%d\n",d_r[index]);
	printf("d_y%d\n",d_y[index]);
	//int buffer = (d_y[index] > X_) ? d_y[index] : ok;
	//printf("buffer:%d\n",buffer);
	//printf("compare:%d\n",buffer > Y

	if(d_y[index] > Y_MAX || d_y[index] <= Y_MIN || d_x[index] >= X_MAX || d_x[index] < X_MIN){
		printf("Err: in arr, coordinate has exceded dimension\n");
                printf("Err: x->%d y->%d\n",d_x[index],d_y[index]);
                printf("Note: actual X_MAX is %d and Y_MIN is %d\n",X_MAX - 1, Y_MIN + 1);
        }else{
		//printf("index:%d\n",index);
                d_y[index] = 0 - d_y[index];
                d_r[(d_y[index] + Y_MAX) * x_lim + d_x[index] + X_MAX] = r;
                d_g[(d_y[index] + Y_MAX) * x_lim + d_x[index] + X_MAX] = g;
                d_b[(d_y[index] + Y_MAX) * x_lim + d_x[index] + X_MAX] = b;

		//printf("arr:%d\n",(y[index] + Y_MAX) * x_lim + x[index] + X_MAX);
		//printf("hello!\n");
        }
}

struct Array array_init(struct Array arr){
	int block = (X_MAX - X_MIN) * (Y_MAX - Y_MIN);
	int size = block * sizeof(int);
	//int * gp_arr_r;
	//int * gp_arr_g;
	//int * gp_arr_b;
	//arr.d_r = (int*)malloc(size);
	//arr.d_g = (int*)malloc(size);
	//arr.d_b = (int*)malloc(size);
	hipMalloc((void **)&(arr.d_r),size);
	hipMalloc((void **)&(arr.d_g),size);
	hipMalloc((void **)&(arr.d_b),size);
	gp_array_list_init<<<block,1>>>(arr.d_r,ARR_R);
	gp_array_list_init<<<block,1>>>(arr.d_g,ARR_G);
	gp_array_list_init<<<block,1>>>(arr.d_b,ARR_B);
//	hipMemcpy(arr.r, gp_arr_r, size, hipMemcpyDeviceToHost);
//	hipMemcpy(arr.g, gp_arr_g, size, hipMemcpyDeviceToHost);
//	hipMemcpy(arr.b, gp_arr_b, size, hipMemcpyDeviceToHost);
//	hipFree(gp_arr_r);
//	hipFree(gp_arr_g);
//	hipFree(gp_arr_b);
	return arr;
}
/*
struct Array arr_set(struct Array arr,int x, int y, int color[3]){
	int x_lim = X_MAX - X_MIN;

	if(y > Y_MAX || y <= Y_MIN || x >= X_MAX || x < X_MIN){
		printf("Err: in arr, coordinate has exceded dimension\n");
		printf("Err: x->%d y->%d\n",x,y);
		printf("Note: actual X_MAX is %d and Y_MIN is %d\n",X_MAX - 1, Y_MIN + 1);
	}else{
			y = 0 - y;
		arr.r[(y + Y_MAX) * x_lim + x + X_MAX] = color[0];
		arr.g[(y + Y_MAX) * x_lim + x + X_MAX] = color[1];
		arr.b[(y + Y_MAX) * x_lim + x + X_MAX] = color[2];
	}

	return arr;
}

/*
void array_print(struct Array arr){
	int size = (X_MAX - X_MIN) * (Y_MAX - Y_MIN);
	int i;
	for(i = 0;i < size; i++){
		printf("r[%d] = %d\n",i,arr.r[i]);
		printf("g[%d] = %d\n",i,arr.g[i]);
		printf("b[%d] = %d\n",i,arr.b[i]);
	}
}

*/
void array_push(struct Array arr, char * filename){
	int y_lim = Y_MAX - Y_MIN;
	int x_lim = X_MAX - X_MIN;

	char line[20];

	int * h_r;
	int * h_g;
	int * h_b;

        int block = (X_MAX - X_MIN) * (Y_MAX - Y_MIN);
        int size = block * sizeof(int);

	h_r = (int*)malloc(size);
	h_g = (int*)malloc(size);
	h_b = (int*)malloc(size);

        hipMemcpy(h_r, arr.d_r, size, hipMemcpyDeviceToHost);
        hipMemcpy(h_g, arr.d_g, size, hipMemcpyDeviceToHost);
        hipMemcpy(h_b, arr.d_b, size, hipMemcpyDeviceToHost);

	int i,j,file;
	file = open(filename, O_CREAT | O_TRUNC | O_WRONLY, 0644);

	snprintf(line, sizeof(line), "P3 %d %d 255\n",x_lim,y_lim);
	write (file, line, strlen(line));
	for(i = 0; i < y_lim; i++){
			for(j = 0; j < x_lim; j++){
				snprintf(line, sizeof(line), "%d %d %d\n", 
					h_r[i * y_lim + j], 
					h_g[i * y_lim + j], 
					h_b[i * y_lim + j]);
				write(file, line, strlen(line));
			} 
		}
}

