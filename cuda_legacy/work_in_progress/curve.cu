#include "hip/hip_runtime.h"
#include "engine.h"

/*
__global__ void gp_add_edge(double * d_grid, 
	    				    double * h_grid,
						    double *x, double *y, double z,
						    double *x1, double *y1,int col1,int col2,int row){
int index = blockIdx.x * blockDim.x + threadIdx.x;

int col_size = col1 + col2;

//
	if((index % col_size) < col1){
		//printf("index:%d\n",index);
		//printf("h_grid-index:%f\n",h_grid[index]);
		//printf("ola:%d\n",(index / col_size * col1) + index % col1);
		//printf("index:%d\n",index % 2);
		d_grid[index] = h_grid[(index / col_size * col1) + index % col1];
	    //d_grid[(index / col1) * col_size + index % col1] = h_grid[index];
	}else{
		//printf("lala:%d\n",index / col_size);

		if(index / col_size == 0){
			
		   if(index % 2 == 0){
  			   d_grid[index] = x[(index - col1) / 2];
		   }else{
		   	   d_grid[index] = x1[(index - col1) /2];
		   }
	
		}else if(index / col_size == 1){
		   if(index % 2 == 0){
  			   d_grid[index] = y[(index - col1) / 2];
		   }else{
		   	   d_grid[index] = y1[(index - col1) /2];
		   }
		}else if(index / col_size == 2){
			d_grid[index] = z;
		}else{
			d_grid[index] = 1;
		}
	}
//printf("index:%d\n",index);
}

struct Edge add_circle(struct Edge mx, double x, double y, double z, double r){
	double t = 0;
	double xx = r * cos(t) + x;
	double yy = r * sin(t) + y;
	//t += t_step * (2 * M_PI);
	double * x1;
	double * y1;
	double * x2;
	double * y2;

	int i;
	while(t <= (2 * M_PI) + t_step){
		double xxx = r * cos(t) + x;
		double yyy = r * sin(t) + y;
		//mx = ed_adde(mx,xx,yy,z,xxx,yyy,z);
		x1[i] = xx;
		y1[i] = yy;
		x2[i] = xxx;
		y2[i] = yyy;


		xx = xxx;
		yy = yyy;
		t = t + t_step * (2 * M_PI);
	}
	return mx;
}

/*I give up
struct Matrix bezier(struct Matrix mx, double x1, double y1, double x2, double y2, double x3. double y3, double x4, double y4){
	double t = 0;
	struct Matrix curve;
	curve = mx_init(4,4);
	mx_set(curve,0,0,1);
	mx_set(curve,1,3,1);
	mx_set(curve,2,0,-3);
	mx_set(curve,2,1,3);
	mx_set(curve,3,2,-3);
	mx_set(curve,3,3,3);

	
}
*/

/*
// The stupid way to do the bezier
struct Edge add_bezier(struct Edge mx, double x1, double y1, double x2, double y2, 
									   double x3, double y3, double x4, double y4){
	
	//printf("mx.col:%d\n",mx.col);

	double t = 0;
	int xx = x1;
	int yy = y1;
	double size = (1 + 2 * t_step)/t_step;

	//printf("size:%f\n",size);

	double d_size = size * sizeof(double);
	double col_size = mx.col * sizeof(double);
	int grid_size = (mx.row * mx.col) * sizeof(double);


	double * h_x;
	double * h_y;
	double * h_x1;
	double * h_y1;

	h_x = (double *)malloc(d_size);
	h_y = (double *)malloc(d_size);
	h_x1 = (double *)malloc(d_size);
	h_y1 = (double *)malloc(d_size);

	
	double * d_x;
	double * d_y;
	double * d_x1;
	double * d_y1;
	double * d_grid;
	double * h_grid;

	hipMalloc((void **)&d_x,d_size);
	hipMalloc((void **)&d_y,d_size);
	hipMalloc((void **)&d_x1,d_size);
	hipMalloc((void **)&d_y1,d_size);
	hipMalloc((void **)&h_grid,grid_size);
	hipMalloc((void **)&d_grid,(mx.col + 2 * size) * (mx.row) * sizeof(double));

	int i = 0;
	int malloc_size = (mx.col + size) * (mx.row) * sizeof(double);

	while(t <= 1 + t_step){
		double xxx = (1-t) * ((1-t) * ((1-t) * x1 + t * x2) + t * ((1-t) * x2 + t * x3)) + t * ((1-t) * ((1-t) * x2 + t * x3) + t * ((1-t) * x3 + t * x4));
		double yyy = (1-t) * ((1-t) * ((1-t) * y1 + t * y2) + t * ((1-t) * y2 + t * y3)) + t * ((1-t) * ((1-t) * y2 + t * y3) + t * ((1-t) * y3 + t * y4));
		//mx = ed_adde(mx,xx,yy,0,xxx,yyy,0);
		
		h_x[i] = xx;
		h_y[i] = yy;
		h_x1[i] = xxx;
		h_y1[i] = yyy;
		
		printf("h_x:%f\n",h_x[i]);
		printf("h_x1:%f\n",h_x1[i]);
		xx = xxx;
		yy = yyy;
		t = t + t_step;
		i++;
	}

    hipMemcpy(d_x,h_x,d_size,hipMemcpyHostToDevice);
    hipMemcpy(d_y,h_y,d_size,hipMemcpyHostToDevice);
    hipMemcpy(d_x1,h_x1,d_size,hipMemcpyHostToDevice);
    hipMemcpy(d_y1,h_y1,d_size,hipMemcpyHostToDevice);      
    hipMemcpy(h_grid,mx.grid,grid_size,hipMemcpyHostToDevice);
    //hipMemcpy(d_grid,,col_size,hipMemcpyHostToDevice);

    gp_add_edge<<<mx.col + 2 * size,mx.row>>>(d_grid,h_grid,
    									  d_x, d_y, 0,
						    			  d_x1, d_y1,mx.col,2 * size,mx.row);

    ed_free(mx);
    mx = ed_init(mx,mx.row,mx.col + (2 * size));

    hipDeviceSynchronize();

    hipMemcpy(mx.grid,d_grid,malloc_size,hipMemcpyDeviceToHost);
    	ed_print(mx);
    //ed_print(mx);

	free(h_x);
	free(h_y);
	free(h_x1);
	free(h_y1);

	hipFree(d_x);
	hipFree(d_x1);
	hipFree(d_y);
	hipFree(d_y1);
	hipFree(d_grid);
	hipFree(h_grid);

	return mx;
}
//*/

/*
struct Edge add_hermite(struct Edge mx, double x1, double y1, double x2, double y2, double x3, double y3, double x4, double y4){
	double t = t_step;
	double xx = x1;
	double yy = y1;
	while(t <= 1 + t_step){
		double xxx = x1 * (2 * pow(t,3) - 3 * pow(t,2) + 1) + x3 * (pow(t,3) -pow(t,2)) + x2 * (pow(t,3) - 2 * pow(t,2) + t) + x4 * (3 * pow(t,2) - 2 * pow(t,3));
		double yyy = y1 * (2 * pow(t,3) - 3 * pow(t,2) + 1) + y3 * (pow(t,3) -pow(t,2)) + y2 * (pow(t,3) - 2 * pow(t,2) + t) + y4 * (3 * pow(t,2) - 2 * pow(t,3));
		mx = ed_adde(mx,xx,yy,0,xxx,yyy,0);
		xx = xxx;
		yy = yyy;
		t += t_step;
	}
	return mx;
}
*/
