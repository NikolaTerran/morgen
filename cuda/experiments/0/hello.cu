
#include <hip/hip_runtime.h>
#include <stdio.h>

//casual 50 million interger addition
#define N 50000000

__global__ void mykernel(void){
	
}

__global__ void add(int *a, int *b, int *c){
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int* x, int size)
{
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%N;
	}
}

int main(void){
	int *a,*b,*c;					//host copies of a,b,c
	
	int *d_a,*d_b,*d_c;         //device copies of a,b,c
	
	int size = N * sizeof(int) * 2;

	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);

	//Alloc space for host copies of a,b,c and setup input values
	a = (int *)malloc(size); random_ints(a,N); 
	b = (int *)malloc(size); random_ints(b,N);
	c = (int *)malloc(size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

	add<<<N,1>>>(d_a,d_b,d_c);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	int i;
	for(i = 0; i < N;i++){
		printf("a[%d]: %d\n",i,a[i]);
		printf("b[%d]: %d\n",i,b[i]);
		printf("c[%d]: %d\n",i,c[i]);
	}

	return 0;
}