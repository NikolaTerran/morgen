#include "hip/hip_runtime.h"
#include "engine.h"

/*random side note
	-maintain a stack of coordinate systems
	-changes to our graphics engine
	  -create a c.s. stack
	-transformation will be applied to the top of the stack
	-drawing/adding shapes
	  -generate polygons/edges
	  -appy top c.s. to those points
	  -draw these polygons/edges
	  -clear polygon/edge matrix

*/
int main(void){
	struct Array arr1;
	arr1 = array_init(arr1);
	struct Array arr2;
	arr2 = array_init(arr2);

	int * x_1;
	int * y_1;
	x_1 = (int *)malloc(3 * sizeof(int));
        y_1 = (int *)malloc(3 * sizeof(int));

	x_1[0] = 1;
	x_1[1] = 2;
	x_1[2] = 3;

	y_1[0] = 1;
	y_1[1] = 2;
	y_1[2] = 3;
	
	int *dx;
	int *dy;
/*	
	hipMalloc((void **)&dx,3 * sizeof(int));
        hipMalloc((void**)& dy,3 * sizeof(int));

	hipMemcpy(dx,x_1,3 * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dy,y_1,3 * sizeof(int),hipMemcpyHostToDevice);
*/
	gp_array_set<<<1,3>>>(arr1.d_r,arr1.d_g,arr1.d_b,dx,dy,255,0,0);

/*
	struct Edge ed;
	ed = ed_init(ed,4,0);

	int color[3];
	color[0] = 0;
	color[1] = 0;
    color[2] = 0;

//    ed = ed_adde(ed,75,100,0,99,-249,0);
//    ed = ed_adde(ed,0,0,0,249,0,0);
//    ed = ed_adde(ed,0,0,0,-250,0,0);
/*    ed = ed_adde(ed,0,0,0,0,250,0);

    ed = ed_adde(ed,0,0,0,249,250,0);
    ed = ed_adde(ed,0,0,0,249,-249,0);
    ed = ed_adde(ed,0,0,0,-249,250,0);
    ed = ed_adde(ed,0,0,0,-250,-249,0);

    ed = add_bezier(ed,0,0,100,200,200,100,249,250);
    ed = add_bezier(ed,0,0,-100,200,-200,100.1,-250,250);
//runtime calculation cpu vs gpu

	clock_t cp_start, cp_end, gp_start,gp_end;
	double cpu_time_used, gpu_time_used;
	

	gp_start = clock();
	    arr1 = drawLine(arr1,ed,color);
	gp_end = clock();
	gpu_time_used = ((double) (gp_end - gp_start)) / CLOCKS_PER_SEC;

	cp_start = clock();
	    arr2 = cp_drawLine(arr2,ed,color);
	cp_end = clock();
	cpu_time_used = ((double) (cp_end - cp_start)) / CLOCKS_PER_SEC;

   printf("cpu used: %f seconds\ngpu used: %f seconds\n",cpu_time_used,gpu_time_used);
*/
	array_push(arr1,"gpu.ppm");
//	array_push(arr2,"cpu.ppm");

	return 0;
}
