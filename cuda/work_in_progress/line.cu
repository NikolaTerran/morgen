#include "hip/hip_runtime.h"
#include "engine.h"

__global__ void add(int *a, int *b, int *c){
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

__global__ void gp_drawline(int *x, int *y, int *x1, int *y1, int ** x2, int ** y2){

}

void random_ints(int* x, int size)
{
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%10;
	}
}

void drawLine(struct Array arr, struct Matrix mx, int color[3]){
	int octants;
	int * x;
	int * y;
	int * x1;
	int * y1;
	
	

	int **x2;
	int **y2;				//host copies of a,b,c
	int **d_x2;
	int **d_y2;         //device copies of a,b,c
	
	int size = N * sizeof(int);

	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);

	hipMalloc((void **)&d_c,size);
	if(mx == x1 && y2 == y1){
				arr. = color[0];
				array[y1][x1][1] = color[1];
				array[y1][x1][2] = color[2];
	}else{
			//{double m = 1;}
		double m = ((double)y1 - (double)y2) / ((double)x2 - (double)x1);
		//db("m",m);
		
			if(y2 <= y1){
				if(x2 >= x1){
					if( m <= 1 ){
						 octants = 1;
					}else{
						 octants = 2;
					}
				}else{
					if( m <= -1 ){
						 octants = 3;
					}else{
						 octants = 4;
					}
				}
			}else{
			   if(x2 < x1){
					if( m <= 1 ){
						 octants = 5;
					}else{
						 octants = 6;
					}
				}else{
					if( m <= -1 ){
						 octants = 7;
					}else{
						 octants = 8;
					}
				}
			}
			
			//db("octants",(double)octants);
			
			array[y1][x1][0] = color[0];
			array[y1][x1][1] = color[1];
			array[y1][x1][2] = color[2];
			//db("lol",(double)octants);
		switch(octants){
				case 1: line_helper(array,x1,y1,x2,y2,x1 + 1,y1 - 1,octants,color,m);
				break;
				case 2: line_helper(array,x1,y1,x2,y2,x1 + 1,y1 - 1,octants,color,m);
				break;
				case 3: line_helper(array,x1,y1,x2,y2,x1 - 1,y1 - 1,octants,color,m);
				break;
				case 4: line_helper(array,x1,y1,x2,y2,x1 - 1,y1 - 1,octants,color,m);//left up
				break;
				case 5: line_helper(array,x2,y2,x1,y1,x2 + 1,y2 - 1,1,color,m);//left down
				break;
				case 6: line_helper(array,x2,y2,x1,y1,x2 + 1,y2 - 1,2,color,m);//down left
				break;
				case 7: line_helper(array,x2,y2,x1,y1,x2 - 1,y2 - 1,3,color,m);//down right
				break;
				case 8: line_helper(array,x2,y2,x1,y1,x2 - 1,y2 - 1,4,color,m);// right down
				break;
		}
	}
}


int main(void){
	int *a,*b,*c;					//host copies of a,b,c
	
	int *d_a,*d_b,*d_c;         //device copies of a,b,c
	
	int size = N * sizeof(int);

	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);

	//Alloc space for host copies of a,b,c and setup input values
	a = (int *)malloc(size); random_ints(a,N); 
	b = (int *)malloc(size); random_ints(b,N);
	c = (int *)malloc(size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

	add<<<N,1>>>(d_a,d_b,d_c);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	int i;
	for(i = 0; i < N;i++){
		printf("a[%d]: %d\n",i,a[i]);
		printf("b[%d]: %d\n",i,b[i]);
		printf("c[%d]: %d\n",i,c[i]);
	}

	return 0;
}


void line_helper(int array[][500][3], int x1, int y1, int x2, int y2, int x3, int y3, int octants, int color[], double m){
	
	double slope;
		switch(octants){
			case 1: 
			while(x3 <= x2){
					slope = ((double)y1 - (double)y3)/((double)x3 - (double)x1);
					if(slope > m){
						//db("slope",slope);
					    //db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 ++;
						//y3 --;
					}else{
					    //db("slope",slope);
					    //db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 ++;
						y3 --;
					}
			}		
			break;
			case 2: //up right
			while(y3 >= y2){
					slope = ((double)y1 - (double)y3)/((double)x3 - (double)x1);
					if(slope > m){
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 ++;
						y3 --;
					}else{
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						//x3 ++;
						y3 --;
					}	
			}	
			break;
			case 3: //up left
			while(y3 >= y2){
					slope = ((double)y1 - (double)y3)/((double)x3 - (double)x1);
					if(slope < m){
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 --;
						y3 --;
					}else{
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						//x3 --;
						y3 --;
					}
			}
			break;
			case 4: //left up
			while(x3 >= x2){
					slope = ((double)y1 - (double)y3)/((double)x3 - (double)x1);
					if(slope < m){
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						
						x3 --;
					}else{
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						y3 --;
						x3 --;
					}
			}
			break;
			case 5: //left down
					if(slope > m){
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 --;
					}else{
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 --;
						y3 ++;
					}
			break;
			case 6: //down left
					if(slope < m){
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						y3 ++;
					}else{
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 --;
						y3 ++;
					}
			break;
			case 7: //down right
					if(slope > m){
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						y3 ++;
						
					}else{
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 ++;
						y3 ++;
					}
			break;
			case 8: //right down
					if(slope < m){
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 ++;
					}else{
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 ++;
						y3 ++;
					}
			break;
		}
	}


