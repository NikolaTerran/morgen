#include "hip/hip_runtime.h"
#include "engine.h"

__global__ void gp_drawline(int *x, int *y, int *x1, int *y1, int * x2, int * y2){

}

struct Array DLMA(
    struct Array arr,
    int x,
    int y,
    int dx,
    int dy,
    char axis,
    int color[3]){
    // calculate some constants
    int dx2 = dx * 2;
    int dy2 = dy * 2;
    int dy2Mindx2 = dy2 - dx2;
 
    // calculate the starting error value
    int Error = dy2 - dx;
 
    // draw the first pixel
    arr = arr_set(arr,x,y,color);
 
    // loop across the major axis
    if(axis == 'x'){
	    while (dx--)
	    {
	        // move on major axis and minor axis
	        if (Error > 0)
	        {
	            x++;
	            y++;
	            Error += dy2Mindx2;
	        }
	        // move on major axis only
	        else
	        {
	            x++;
	            Error += dy2;
	        }
	 
	        // draw the next pixel
	        arr = arr_set(arr,x,y,color);
	    }
	}else{
	    while (dx--)
	    {
	        // move on major axis and minor axis
	        if (Error > 0)
	        {
	            x++;
	            y++;
	            Error += dy2Mindx2;
	        }
	        // move on major axis only
	        else
	        {
	            y++;
	            Error += dy2;
	        }
	 
	        // draw the next pixel
	        arr = arr_set(arr,x,y,color);
	    }
	}
	return arr;
}
 
 
// Specialized Line Drawing optimized for horizontal or vertical lines
// X and Y are flipped for Y maxor axis lines, but the pixel writes are handled correctly due to
// minor and major axis pixel movement
struct Array DLSA(struct Array arr, int x, int y, int dx, char axis, int color[3]){
    // draw the first pixel
 	arr = arr_set(arr,x,y,color);
    // loop across the major axis and draw the rest of the pixels
    if(axis == 'x'){
	    while (dx--){
	    	x++;
        	arr = arr_set(arr,x,y,color);
    	}
    }else{
	    while (dx--){
	    	y++;
        	arr = arr_set(arr,x,y,color);
    	}
    }

    return arr;
}
 
// Draw an arbitrary line.  Assumes start and end point are within valid range
// pixels is a pointer to where the pixels you want to draw to start aka (0,0)
// pixelStride is the number of unsigned ints to get from one row of pixels to the next.
// Usually, that is the same as the width of the image you are drawing to, but sometimes is not.
struct Array DL(struct Array arr, int x1, int y1, int x2, int y2, int color[3]){
    // calculate our deltas
    int dx = x2 - x1;
    int dy = y2 - y1;

    // if the X axis is the major axis
    if (abs(dx) >= abs(dy)){
        // if x2 < x1, flip the points to have fewer special cases
        if (dx < 0){
            dx *= -1;
            dy *= -1;
            int t = x1;
            x1 = x2;
            x2 = t;

            t = y1;
            y1 = y2;
            y2 = t;
        }
 
        // determine special cases
        if(dy > 0){
            arr = DLMA(arr, x1,y1, dx, dy,'x', color);
        }
        else if (dy < 0){
         	arr = DLMA(arr, x1,y1, dx, -dy,'x', color);
        }
        else{
            arr = DLSA(arr,x1,y1,dx,'x', color);
        }
    }
    // else the Y axis is the major axis
    else
    {
        // if y2 < y1, flip the points to have fewer special cases
        if (dy < 0)
        {
            dx *= -1;
            dy *= -1;
            int t = x1;
            x1 = x2;
            x2 = t;

            t = y1;
            y1 = y2;
            y2 = t;
        }
 
        // get the address of the pixel at (x1,y1)
 
        // determine special cases
        if (dx > 0){
         	  arr = DLMA(arr, x1,y1, dy, dx,'y', color);
        }else if (dx < 0){
        	  arr = DLMA(arr, x1,y1, dy, -dx,'y', color);
        }else{
              arr = DLSA(arr,x1,y1, dy,'y', color);
        }
    }

    return arr;
}


/*
void drawLine(struct Array arr, struct Matrix mx, int color[3]){
	int octants;
	int * x;
	int * y;
	int * x1;
	int * y1;
	int * x2;
	int * y2;

	x = mx_toint(mx,x,0,0);
	y = mx_toint(mx,y,1,0);
	x1 = mx_toint(mx,x1,0,1);
	y1 = mx_toint(mx,x1,1,1);

	int * d_x;
	int * d_y;
	int * d_x1;
	int * d_y1;
	int * d_x2;
	int * d_y2;

	hipMalloc((void **)&d_x,mx.num_edge);
	hipMalloc((void **)&d_y,mx.num_edge);
	hipMalloc((void **)&d_x1,mx.num_edge);
	hipMalloc((void **)&d_y1,mx.num_edge);
	//hipMalloc((void **)&d_y1,mx.num_edge);

        hipMemcpy(d_x,x,mx.num_edge,hipMemcpyHostToDevice);
        hipMemcpy(d_y,y,mx.num_edge,hipMemcpyHostToDevice);
        hipMemcpy(d_x1,x1,mx.num_edge,hipMemcpyHostToDevice);
        hipMemcpy(d_y1,y1,mx.num_edge,hipMemcpyHostToDevice);


}


/*

	int * x2;
	int * y2;	    //host copies of a,b,c
	int * d_x2;
	int * d_y2;         //device copies of a,b,c
	
	int size = N * sizeof(int);

	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);

	hipMalloc((void **)&d_c,size);
	if(mx == x1 && y2 == y1){
				arr. = color[0];
				array[y1][x1][1] = color[1];
				array[y1][x1][2] = color[2];
	}else{
			//{double m = 1;}
		double m = ((double)y1 - (double)y2) / ((double)x2 - (double)x1);
		//db("m",m);
		
			if(y2 <= y1){
				if(x2 >= x1){
					if( m <= 1 ){
						 octants = 1;
					}else{
						 octants = 2;
					}
				}else{
					if( m <= -1 ){
						 octants = 3;
					}else{
						 octants = 4;
					}
				}
			}else{
			   if(x2 < x1){
					if( m <= 1 ){
						 octants = 5;
					}else{
						 octants = 6;
					}
				}else{
					if( m <= -1 ){
						 octants = 7;
					}else{
						 octants = 8;
					}
				}
			}
			
			//db("octants",(double)octants);
			
			array[y1][x1][0] = color[0];
			array[y1][x1][1] = color[1];
			array[y1][x1][2] = color[2];
			//db("lol",(double)octants);
		switch(octants){
				case 1: line_helper(array,x1,y1,x2,y2,x1 + 1,y1 - 1,octants,color,m);
				break;
				case 2: line_helper(array,x1,y1,x2,y2,x1 + 1,y1 - 1,octants,color,m);
				break;
				case 3: line_helper(array,x1,y1,x2,y2,x1 - 1,y1 - 1,octants,color,m);
				break;
				case 4: line_helper(array,x1,y1,x2,y2,x1 - 1,y1 - 1,octants,color,m);//left up
				break;
				case 5: line_helper(array,x2,y2,x1,y1,x2 + 1,y2 - 1,1,color,m);//left down
				break;
				case 6: line_helper(array,x2,y2,x1,y1,x2 + 1,y2 - 1,2,color,m);//down left
				break;
				case 7: line_helper(array,x2,y2,x1,y1,x2 - 1,y2 - 1,3,color,m);//down right
				break;
				case 8: line_helper(array,x2,y2,x1,y1,x2 - 1,y2 - 1,4,color,m);// right down
				break;
		}
	}
}


int main(void){
	int *a,*b,*c;		    //host copies of a,b,c
	
	int *d_a,*d_b,*d_c;         //device copies of a,b,c
	
	int size = N * sizeof(int);

	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);

	//Alloc space for host copies of a,b,c and setup input values
	a = (int *)malloc(size); random_ints(a,N); 
	b = (int *)malloc(size); random_ints(b,N);
	c = (int *)malloc(size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

	add<<<N,1>>>(d_a,d_b,d_c);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	int i;
	for(i = 0; i < N;i++){
		printf("a[%d]: %d\n",i,a[i]);
		printf("b[%d]: %d\n",i,b[i]);
		printf("c[%d]: %d\n",i,c[i]);
	}

	return 0;
}


void line_helper(int array[][500][3], int x1, int y1, int x2, int y2, int x3, int y3, int octants, int color[], double m){
	
	double slope;
		switch(octants){
			case 1: 
			while(x3 <= x2){
					slope = ((double)y1 - (double)y3)/((double)x3 - (double)x1);
					if(slope > m){
						//db("slope",slope);
					    //db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 ++;
						//y3 --;
					}else{
					    //db("slope",slope);
					    //db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 ++;
						y3 --;
					}
			}		
			break;
			case 2: //up right
			while(y3 >= y2){
					slope = ((double)y1 - (double)y3)/((double)x3 - (double)x1);
					if(slope > m){
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 ++;
						y3 --;
					}else{
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						//x3 ++;
						y3 --;
					}	
			}	
			break;
			case 3: //up left
			while(y3 >= y2){
					slope = ((double)y1 - (double)y3)/((double)x3 - (double)x1);
					if(slope < m){
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 --;
						y3 --;
					}else{
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						//x3 --;
						y3 --;
					}
			}
			break;
			case 4: //left up
			while(x3 >= x2){
					slope = ((double)y1 - (double)y3)/((double)x3 - (double)x1);
					if(slope < m){
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						
						x3 --;
					}else{
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						y3 --;
						x3 --;
					}
			}
			break;
			case 5: //left down
					if(slope > m){
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 --;
					}else{
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 --;
						y3 ++;
					}
			break;
			case 6: //down left
					if(slope < m){
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						y3 ++;
					}else{
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 --;
						y3 ++;
					}
			break;
			case 7: //down right
					if(slope > m){
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						y3 ++;
						
					}else{
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 ++;
						y3 ++;
					}
			break;
			case 8: //right down
					if(slope < m){
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 ++;
					}else{
						//db("slope",slope);
						//db("y3",y3);
						//db("x3",x3);
						array[y3][x3][0] = color[0];
						array[y3][x3][1] = color[1];
						array[y3][x3][2] = color[2];
						x3 ++;
						y3 ++;
					}
			break;
		}
	}


*/
