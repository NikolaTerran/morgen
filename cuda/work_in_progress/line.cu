#include "hip/hip_runtime.h"
#include "engine.h"

__global__ void gp_drawline(int *r, int *g, int *b, int *d_x, int *d_y, int *d_x1, int *d_y1){
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int dx = d_x1[index] - d_x[index];
    int dy = d_y1[index] - d_y[index];

    // if the X axis is the major axis
    if (abs(dx) >= abs(dy)){
        // if x2 < x1, flip the points to have fewer special cases
        if(dx < 0){
            dx *= -1;
            dy *= -1;
            int t = d_x[index];
            d_x[index] = d_x1[index];
            d_x1[index] = t;

            t = d_y[index];
            d_y[index] = d_y1[index];
            d_y1[index] = t;
        }
 
        // determine special cases
        if(dy > 0){
            arr = DLMA(arr, x1,y1, dx, dy, 0,'x', color);
        }
        else if (dy < 0){
         	arr = DLMA(arr, x1,y1, dx, -dy, 1,'x', color);
        }
        else{
            arr = DLSA(arr,x1,y1,dx,'x', color);
        }
    }
    // else the Y axis is the major axis
    else
    {
        // if y2 < y1, flip the points to have fewer special cases
        if (dy < 0)
        {
            dx *= -1;
            dy *= -1;
            int t = x1;
            x1 = x2;
            x2 = t;

            t = y1;
            y1 = y2;
            y2 = t;
        }
 
        // get the address of the pixel at (x1,y1)
 
        // determine special cases
        if (dx > 0){
         	  arr = DLMA(arr, x1,y1, dy, dx, 0,'y', color);
        }else if (dx < 0){
        	  arr = DLMA(arr, x1,y1, dy, -dx, 1,'y', color);
        }else{
              arr = DLSA(arr,x1,y1, dy,'y', color);
        }
    }
}

struct Array DLMA(
    struct Array arr,
    int x,
    int y,
    int dx,
    int dy,
    int check,
    char axis,
    int color[3]){
    // calculate some constants
    int dx2 = dx * 2;
    int dy2 = dy * 2;
    int dy2Mindx2 = dy2 - dx2;
 
    // calculate the starting error value
    int Error = dy2 - dx;
 
    // draw the first pixel
    arr = arr_set(arr,x,y,color);
 
    // loop across the major axis
    if(axis == 'x'){
	    while (dx--)
	    {
	        // move on major axis and minor axis
	        if (Error > 0)
	        {
	            x++;
		    if(check == 0){
		            y++;
		    }else{
			y--;
		    }
	            Error += dy2Mindx2;
	        }
	        // move on major axis only
	        else
	        {
	            x++;
	            Error += dy2;
	        }
	 
	        // draw the next pixel
	        arr = arr_set(arr,x,y,color);
	    }
	}else{
	    while (dx--)
	    {
	        // move on major axis and minor axis
	        if (Error > 0)
	        {
			if(check == 0){
	            x++;}else{x--;}
	            y++;
	            Error += dy2Mindx2;
	        }
	        // move on major axis only
	        else
	        {
	            y++;
	            Error += dy2;
	        }
	 
	        // draw the next pixel
	        arr = arr_set(arr,x,y,color);
	    }
	}
	return arr;
}


// Specialized Line Drawing optimized for horizontal or vertical lines
// X and Y are flipped for Y maxor axis lines, but the pixel writes are handled correctly due to
// minor and major axis pixel movement
struct Array DLSA(struct Array arr, int x, int y, int dx, char axis, int color[3]){
    // draw the first pixel
 	arr = arr_set(arr,x,y,color);
    // loop across the major axis and draw the rest of the pixels


    if(axis == 'x'){
	    while (dx--){
	    	x++;
        	arr = arr_set(arr,x,y,color);
    	}
    }else{
	    while (dx--){
	    	y++;
        	arr = arr_set(arr,x,y,color);
    	}
    }

    return arr;
}
 
// Draw an arbitrary line.  Assumes start and end point are within valid range
// pixels is a pointer to where the pixels you want to draw to start aka (0,0)
// pixelStride is the number of unsigned ints to get from one row of pixels to the next.
// Usually, that is the same as the width of the image you are drawing to, but sometimes is not.
struct Array DL(struct Array arr, int x1, int y1, int x2, int y2, int color[3]){
    // calculate our deltas
    int dx = x2 - x1;
    int dy = y2 - y1;


    // if the X axis is the major axis
    if (abs(dx) >= abs(dy)){
        // if x2 < x1, flip the points to have fewer special cases
        if (dx < 0){
            dx *= -1;
            dy *= -1;
            int t = x1;
            x1 = x2;
            x2 = t;

            t = y1;
            y1 = y2;
            y2 = t;
        }
 
        // determine special cases
        if(dy > 0){
            arr = DLMA(arr, x1,y1, dx, dy, 0,'x', color);
        }
        else if (dy < 0){
         	arr = DLMA(arr, x1,y1, dx, -dy, 1,'x', color);
        }
        else{
            arr = DLSA(arr,x1,y1,dx,'x', color);
        }
    }
    // else the Y axis is the major axis
    else
    {
        // if y2 < y1, flip the points to have fewer special cases
        if (dy < 0)
        {
            dx *= -1;
            dy *= -1;
            int t = x1;
            x1 = x2;
            x2 = t;

            t = y1;
            y1 = y2;
            y2 = t;
        }
 
        // get the address of the pixel at (x1,y1)
 
        // determine special cases
        if (dx > 0){
         	  arr = DLMA(arr, x1,y1, dy, dx, 0,'y', color);
        }else if (dx < 0){
        	  arr = DLMA(arr, x1,y1, dy, -dx, 1,'y', color);
        }else{
              arr = DLSA(arr,x1,y1, dy,'y', color);
        }
    }

    return arr;
}



struct Array drawLine(struct Array arr, struct Edge ed, int color[3]){
	int * x;
	int * y;
	int * x1;
	int * y1;

	x = ed_toint(ed,x,0,0);
	y = ed_toint(ed,y,1,0);
	x1 = ed_toint(ed,x1,0,1);
	y1 = ed_toint(ed,x1,1,1);

/*
	int * d_x;
	int * d_y;
	int * d_x1;
	int * d_y1;
	int * d_x2;
	int * d_y2;
/*
	hipMalloc((void **)&d_x,mx.num_edge);
	hipMalloc((void **)&d_y,mx.num_edge);
	hipMalloc((void **)&d_x1,mx.num_edge);
	hipMalloc((void **)&d_y1,mx.num_edge);
	//hipMalloc((void **)&d_y1,mx.num_edge);

        hipMemcpy(d_x,x,mx.num_edge,hipMemcpyHostToDevice);
        hipMemcpy(d_y,y,mx.num_edge,hipMemcpyHostToDevice);
        hipMemcpy(d_x1,x1,mx.num_edge,hipMemcpyHostToDevice);
        hipMemcpy(d_y1,y1,mx.num_edge,hipMemcpyHostToDevice);
*/
	int i;
	for(i = 0; i < ed.col/2; i++){
		printf("x: %d  y: %d  x1: %d  y1: %d\n",x[i],y[i],x1[i],y1[i]);
		arr = DL(arr,x[i],y[i],x1[i],y1[i],color);
	}
	return arr;
}


/*
int main(void){
	int *a,*b,*c;		    //host copies of a,b,c
	
	int *d_a,*d_b,*d_c;         //device copies of a,b,c
	
	int size = N * sizeof(int);

	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);

	//Alloc space for host copies of a,b,c and setup input values
	a = (int *)malloc(size); random_ints(a,N); 
	b = (int *)malloc(size); random_ints(b,N);
	c = (int *)malloc(size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

	add<<<N,1>>>(d_a,d_b,d_c);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	int i;
	for(i = 0; i < N;i++){
		printf("a[%d]: %d\n",i,a[i]);
		printf("b[%d]: %d\n",i,b[i]);
		printf("c[%d]: %d\n",i,c[i]);
	}

	return 0;
}
*/
