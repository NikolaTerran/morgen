#include "hip/hip_runtime.h"
#include "engine.h"

/////////////////////////////
//basic matrix manipulation//
/////////////////////////////

void mx_print(struct Matrix mx){
	int i,j;
	for(i = 0; i < mx.row; i++){
		printf("[  ");
		for(j = 0; j < mx.col; j++){
			printf("%0.2f  ",mx.grid[i * mx.col + j]);
		}
		printf("]\n");
	}
}

int * mx_rowtoint( struct Matrix mx, int * ptr,int row){
	ptr = (int *)malloc(mx.col * sizeof(int));
	int i;
	for(i = 0; i < mx.col; i++){
		ptr[i] = mx_get(mx,row,i);
	}
	return ptr;
}

double mx_get(struct Matrix mx, int row, int col){
	int i, j;
	for(i = 0; i <= mx.row; i++){
		for(j = 0; j <= mx.col; j++){
			if(i == row && j == col){
				return mx.grid[i * mx.col + col];
			}
		}
	}
}

struct Matrix mx_init(struct Matrix mx, int row, int col){
	mx.row = row;
	mx.col = col;

	mx.grid = (double *)malloc(row * col * sizeof(double));

	int i, j;

	i = row;
	j = col;
	for(i = 0; i < row; i++){
		for(j = 0; j < col; j++){
			mx.grid[i * col + j] = INIT_VALUE;
		}
	}
	return mx;
}

struct Matrix mx_set(struct Matrix mx, int row, int col, double val){
	if(row >= mx.row || col >= mx.col){
    	printf("ERROR: mx_set, target row or column doesn't exist, return original Matrix\n");
    	return mx;
    }
		int i, j;
		for(i = 0; i <= row; i++){
			for(j = 0; j <= col; j++){	
				if(j == col && i == row){
					mx.grid[i * mx.col + j] = val;
				}
			}
		}
	return mx;
}

struct Matrix mx_addc(struct Matrix m1, struct Matrix m2){
	struct Matrix m3;
	if(m1.row != m2.row){
		printf("ERROR: mx_addc, two matrices have different numbers of rows, return an empty matrix\n");
		return m3;
	}else{
	
		int column = m1.col + m2.col;
		m3 = mx_init(m3,m1.row,column);
		int i, j;
		for(i = 0; i < m3.row; i++){
			for(j = 0; j < m3.col; j++){
			
				if(j < m1.col){
					m3 = mx_set(m3,i,j,m1.grid[i * m1.col + j]);
				}else{
					m3 = mx_set(m3,i,j,m2.grid[i * m2.col + j - m1.col]);
				}	
			}
			j = 0;
		}
		return m3;
	}
}

struct Matrix mx_addp(struct Matrix mx, int x, int y, int z){
	struct Matrix m1;
	m1 = mx_init(m1,4,1);
	m1 = mx_set(m1,0,0,x);
	m1 = mx_set(m1,1,0,y);
	m1 = mx_set(m1,2,0,z);
	m1 = mx_set(m1,3,0,1);

	mx = mx_addc(mx,m1);

	return mx;
}

struct Matrix mx_adde(struct Matrix mx, int x, int y, int z, int x1, int y1, int z1){
	mx = mx_addp(mx, x,y,z);
	mx = mx_addp(mx, x1,y1,z1);

	return mx;
}

void mx_export(struct Matrix mx, struct Array arr){
	
}
