#include "hip/hip_runtime.h"
#include "engine.h"

/////////////////////////////
//basic matrix manipulation//
/////////////////////////////

void ed_print(struct Edge mx){
	int i,j;
	for(i = 0; i < mx.row; i++){
		printf("[  ");
		for(j = 0; j < mx.col; j++){
			printf("%0.2f  ",mx.grid[i * mx.col + j]);
		}
		printf("]\n");
	}
}

int * ed_toint( struct Edge mx, int * ptr,int row,int select){
	if(mx.col == 0){
		printf("Err: ed_toint, edge matrix has 0 column\n");
		exit(1);
	}
	ptr = (int *)malloc(mx.col / 2 * sizeof(int));
	int i,j;
	if(select == 0){
		i = 0;
		j = 0;
	}else{
		i = 1;
		j = 0;
	}
	for(i; i < mx.col; i += 2){
		ptr[j] = ed_get(mx,row,i);
		//printf("lala:%d\n",ptr[j]);
		j++;
	}
	return ptr;
}

double ed_get(struct Edge mx, int row, int col){
	int i, j;
	for(i = 0; i <= mx.row; i++){
		for(j = 0; j <= mx.col; j++){
			if(i == row && j == col){
				return mx.grid[i * mx.col + col];
			}
		}
	}
}

struct Edge ed_init(struct Edge mx, int row, int col){
	mx.row = row;
	mx.col = col;
	//mx.num_edge = 0;

	mx.grid = (double *)malloc(row * col * sizeof(double));

	int i, j;

	i = row;
	j = col;
	for(i = 0; i < row; i++){
		for(j = 0; j < col; j++){
			mx.grid[i * col + j] = INIT_VALUE;
		}
	}
	return mx;
}

struct Edge ed_set(struct Edge mx, int row, int col, double val){
	if(row >= mx.row || col >= mx.col){
    	printf("ERROR: mx_set, target row or column doesn't exist, return original Matrix\n");
    	return mx;
    }
		int i, j;
		for(i = 0; i <= row; i++){
			for(j = 0; j <= col; j++){	
				if(j == col && i == row){
					mx.grid[i * mx.col + j] = val;
				}
			}
		}
	return mx;
}

struct Edge ed_addc(struct Edge m1, struct Edge m2){
	struct Edge m3;
	if(m1.row != m2.row){
		printf("ERROR: mx_addc, two matrices have different numbers of rows, return an empty matrix\n");
		return m3;
	}else{
		int column = m1.col + m2.col;
		m3 = ed_init(m3,m1.row,column);
		int i, j;
		for(i = 0; i < m3.row; i++){
			for(j = 0; j < m3.col; j++){
				if(j < m1.col){
					m3 = ed_set(m3,i,j,m1.grid[i * m1.col + j]);
				}else{
					m3 = ed_set(m3,i,j,m2.grid[i * m2.col + j - m1.col]);
				}
			}
			j = 0;
		}
		return m3;
	}
}

struct Edge ed_addp(struct Edge mx, int x, int y, int z){
	struct Edge m1;
	m1 = ed_init(m1,4,1);
	m1 = ed_set(m1,0,0,x);
	m1 = ed_set(m1,1,0,y);
	m1 = ed_set(m1,2,0,z);
	m1 = ed_set(m1,3,0,1);

	mx = ed_addc(mx,m1);

	return mx;
}

struct Edge ed_adde(struct Edge mx, int x, int y, int z, int x1, int y1, int z1){
	mx = ed_addp(mx, x,y,z);
	mx = ed_addp(mx, x1,y1,z1);
	mx.num_edge = mx.col / 2;

	return mx;
}

void ed_export(struct Edge mx, struct Array arr){

}

