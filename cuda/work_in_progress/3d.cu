#include "hip/hip_runtime.h"
#include "engine.h"


struct Edge add_sphere(struct Edge mx, double x, double y, double z, double r){
	double t = 0;
	struct Edge trans;
	struct Edge result;
	result = ed_init(4,0);
	//t += t_step * (2 * M_PI);
	while(t <= (2 * M_PI) + td_step){
		trans = circle(mx, 0 , 0 , 0 , r);
		trans = mx_rotate(trans,td_axis,t);
		result = mx_addc(result,trans);
		t += td_step;
	//	printf("t: %f\n",t);
	}
	result = mx_transform(result,x,y,z);
	mx = mx_addc(mx,result);
	//mx_print(trans);
	//trans = mx_transform(trans,x,y,z);
	//mx = mx_addc(mx,trans);
	return mx;
}

struct Edge draw_poly(struct Matrix mx){
	return mx;
}

struct Edge add_torus(struct Matrix mx, double x, double y, double z, double r, double d, int axis){
	double t = 0;
	struct Edge trans;
	struct Edge result;
	result = ed_init(4,0);
	//t += t_step * (2 * M_PI);
	double a = 0;
	double b = 0;
	double c = 0;

	if(axis == 0){
		b = d;
	}else if(axis == 1){
		a = d;
	}else{
		c = d;
	}
	while(t <= (2 * M_PI) + td_step){
		trans = add_circle(mx, a , b , c , r);
		trans = mx_rotate(trans,axis,t);
		result = mx_addc(result,trans);
		t += td_step;
	//	printf("t: %f\n",t);
	}
	result = mx_transform(result,x,y,z);
	mx = mx_addc(mx,result);

	return mx;
}

struct Edge add_box(struct Edge mx, double x1, double y1, double z1,
										double h, double l, double w){
	mx = ed_adde(mx,x1    , y1    , z1    , x1 + w, y1    , z1    );
	mx = ed_adde(mx,x1    , y1    , z1    , x1    , y1 + h, z1    );
	mx = ed_adde(mx,x1    , y1    , z1    , x1    , y1    , z1 + l);
	mx = ed_adde(mx,x1    , y1    , z1 + l, x1 + w, y1    , z1 + l);
	mx = ed_adde(mx,x1    , y1    , z1 + l, x1    , y1 + h, z1 + l);
	mx = ed_adde(mx,x1 + w, y1    , z1 + l, x1 + w, y1    , z1	  );
	mx = ed_adde(mx,x1 + w, y1    , z1    , x1 + w, y1 + h, z1    );
	mx = ed_adde(mx,x1    , y1 + h, z1    , x1 + w, y1 + h, z1    );
	mx = ed_adde(mx,x1    , y1 + h, z1 + l, x1    , y1 + h, z1    );
	mx = ed_adde(mx,x1    , y1 + h, z1 + l, x1 + w, y1 + h, z1 + l);
	mx = ed_adde(mx,x1 + w, y1 + h, z1 + l, x1 + w, y1    , z1 + l);
	mx = ed_adde(mx,x1 + w, y1 + h, z1 + l, x1 + w, y1 + h, z1    );

	return mx;

}
