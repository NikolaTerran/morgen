#include "hip/hip_runtime.h"
#include "engine.h"

//using cuda to initiate canvas array

__global__ void gp_array_list_init(int *list, int val){

	list[blockIdx.x] = val;
	//printf("list[%d]:%d\n",blockIdx.x,list[blockIdx.x]);
}

struct Array array_init(struct Array arr){
	int block = (X_MAX - X_MIN) * (Y_MAX - Y_MIN);
	int size = block * sizeof(int);

	int * gp_arr_r;
	int * gp_arr_g;
	int * gp_arr_b;

	arr.r = (int*)malloc(size);
	arr.g = (int*)malloc(size);
	arr.b = (int*)malloc(size);

	hipMalloc((void **)&gp_arr_r,size);
	hipMalloc((void **)&gp_arr_g,size);
	hipMalloc((void **)&gp_arr_b,size);

	gp_array_list_init<<<block,1>>>(gp_arr_r,ARR_R);
	gp_array_list_init<<<block,1>>>(gp_arr_g,ARR_G);
	gp_array_list_init<<<block,1>>>(gp_arr_b,ARR_B);

	hipMemcpy(arr.r, gp_arr_r, size, hipMemcpyDeviceToHost);
	hipMemcpy(arr.g, gp_arr_g, size, hipMemcpyDeviceToHost);
	hipMemcpy(arr.b, gp_arr_b, size, hipMemcpyDeviceToHost);

	hipFree(gp_arr_r);
	hipFree(gp_arr_g);
	hipFree(gp_arr_b);

	return arr; 
}

struct Array arr_set(struct Array arr,int x, int y, int color[3]){
	int x_lim = X_MAX - X_MIN;

	if(y > Y_MAX || y <= Y_MIN || x >= X_MAX || x < X_MIN){
		printf("Err: in arr, coordinate has exceded dimension\n");
		printf("Err: x->%d y->%d\n",x,y);
		printf("Note: actual X_MAX is %d and Y_MIN is %d\n",X_MAX - 1, Y_MIN + 1);
	}else{
			y = 0 - y;
		arr.r[(y + Y_MAX) * x_lim + x + X_MAX] = color[0];
		arr.g[(y + Y_MAX) * x_lim + x + X_MAX] = color[1];
		arr.b[(y + Y_MAX) * x_lim + x + X_MAX] = color[2];
	}

	return arr;
}

void array_print(struct Array arr){
	int size = (X_MAX - X_MIN) * (Y_MAX - Y_MIN);
	int i;
	for(i = 0;i < size; i++){
		printf("r[%d] = %d\n",i,arr.r[i]);
		printf("g[%d] = %d\n",i,arr.g[i]);
		printf("b[%d] = %d\n",i,arr.b[i]);
	}
}

void array_push(struct Array arr, char * filename){
	int y_lim = Y_MAX - Y_MIN;
	int x_lim = X_MAX - X_MIN;

	char line[20];

	int i,j,file;
	file = open(filename, O_CREAT | O_TRUNC | O_WRONLY, 0644);

	snprintf(line, sizeof(line), "P3 %d %d 255\n",x_lim,y_lim);
	write (file, line, strlen(line));
	for(i = 0; i < y_lim; i++){
			for(j = 0; j < x_lim; j++){
				snprintf(line, sizeof(line), "%d %d %d\n", 
					arr.r[i * y_lim + j], 
					arr.g[i * y_lim + j], 
					arr.b[i * y_lim + j]);
				write(file, line, strlen(line));
			}		 
		}
}
